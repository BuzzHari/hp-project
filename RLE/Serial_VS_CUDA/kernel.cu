#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

/* #include "helper_timer.h" */
#include "hip/hip_runtime_api.h"

#include "hemi/grid_stride_range.h"
#include "hemi/launch.h"

#include "chag/pp/prefix.cuh"
#include "chag/pp/reduce.cuh"

#include <iostream>
#include <time.h>

namespace pp = chag::pp;

// global host memory arrays.
int* g_symbolsOut;
int* g_countsOut;
int* g_in;
int* g_decompressed;

// Device memory used in PARLE
int* d_symbolsOut;
int* d_countsOut;
int* d_in;
int* d_totalRuns;
int* d_backwardMask;
int* d_scannedBackwardMask;
int* d_compactedBackwardMask;

const int NUM_TESTS = 4;
const int Tests[NUM_TESTS] = {
    8388608,
    16777216,
    33554432, // 32MB
    67108864 // 64MB
};

const int PROFILING_TESTS = 1;
const int MAX_N = 1 << 26; // max size of any array that we use.

void parleDevice(int *d_in, int n,
    int* d_symbolsOut,
    int* d_countsOut,
    int* d_totalRuns
    );

int parleHost(int *h_in, int n,
    int* h_symbolsOut,
    int* h_countsOut);
int rleCpu(int *in, int n,
    int* symbolsOut,
    int* countsOut);

__global__ void compactKernel(int* g_in, int* g_scannedBackwardMask, int* g_compactedBackwardMask, int* g_totalRuns, int n) {
    for (int i : hemi::grid_stride_range(0, n)) {

        if (i == (n - 1)) {
            g_compactedBackwardMask[g_scannedBackwardMask[i] + 0] = i + 1;
            *g_totalRuns = g_scannedBackwardMask[i];
        }

        if (i == 0) {
            g_compactedBackwardMask[0] = 0;
        }
        else if (g_scannedBackwardMask[i] != g_scannedBackwardMask[i - 1]) {
            g_compactedBackwardMask[g_scannedBackwardMask[i] - 1] = i;
        }
    }
}

__global__ void scatterKernel(int* g_compactedBackwardMask, int* g_totalRuns, int* g_in, int* g_symbolsOut, int* g_countsOut) {
    int n = *g_totalRuns;

    for (int i : hemi::grid_stride_range(0, n)) {
        int a = g_compactedBackwardMask[i];
        int b = g_compactedBackwardMask[i + 1];

        g_symbolsOut[i] = g_in[a];
        g_countsOut[i] = b - a;
    }
}

__global__ void maskKernel(int *g_in, int* g_backwardMask, int n) {
    for (int i : hemi::grid_stride_range(0, n)) {
        if (i == 0)
            g_backwardMask[i] = 1;
        else {
            g_backwardMask[i] = (g_in[i] != g_in[i - 1]);
        }
    }
}

void PrintArray(int* arr, int n){
    for (int i = 0; i < n; ++i){
        printf("%d, ", arr[i]);
    }
    printf("\n");
}

char errorString[256];

bool verifyCompression(
    int* original, int n,
    int* compressedSymbols, int* compressedCounts, int totalRuns){

    // decompress.
    int j = 0;

    int sum = 0;
    for (int i = 0; i < totalRuns; ++i) {
        sum += compressedCounts[i];
    }

    if (sum != n) {
        sprintf(errorString, "Decompressed and original size not equal %d != %d\n", n, sum);

        for (int i = 0; i < totalRuns; ++i){
            int symbol = compressedSymbols[i];
            int count = compressedCounts[i];

            printf("%d, %d\n", count, symbol);
        }
        return false;
    }

    for (int i = 0; i < totalRuns; ++i){
        int symbol = compressedSymbols[i];
        int count = compressedCounts[i];

        for (int k = 0; k < count; ++k){
            g_decompressed[j++] = symbol;
        }
    }

    // verify the compression.
    for (int i = 0; i < n; ++i) {
        if (original[i] != g_decompressed[i]){

            sprintf(errorString, "Decompressed and original not equal at %d, %d != %d\n", i, original[i], g_decompressed[i]);
            return false;
        }
    }

    return true;
}

// get random test data for compression.
// the kind of data generated is like
// 1,1,1,1,4,4,4,4,7,7,7,7,....
// so there's lots of repeated sequences.
int* generateCompressibleRandomData(int n){
    int val = rand() % 10;

    for (int i = 0; i < n; ++i) {
        g_in[i] = val;

        if (rand() % 6 == 0){
            val = rand() % 10;
        }
    }
    return g_in;
}


// get random test data for compression.
// the kind of data generated is like
// 1,5,8,4,2,6,....
// so it's completely random.
int* generateRandomData(int n){
    for (int i = 0; i < n; ++i) {
        g_in[i] = rand() % 10;;

    }
    return g_in;
}


// use f to RLE compresss the data, and then verify the compression.
template<typename F>
void unitTest(int* in, int n, F f, bool verbose)
{
    int totalRuns = f(in, n, g_symbolsOut, g_countsOut);

    if (verbose) {
        printf("n = %d\n", n);
        printf("Original Size  : %d\n", n);
        printf("Compressed Size: %d\n\n", totalRuns * 2);
    }

    if (!verifyCompression(
        in, n,
        g_symbolsOut, g_countsOut, totalRuns)) {
        printf("Failed test %s\n", errorString);
        PrintArray(in, n);

        exit(1);
    }
//    else {
  //      if (verbose)
    //        printf("passed test!\n\n");
    //}
}


// profile some RLE implementation on the CPU.
template<typename F, typename G>
void profileCpu(F rle, G dataGen) {
    for (int i = 0; i < NUM_TESTS; ++i) {
        int n = Tests[i];
        //int* in = dataGen(n);
        int* in = dataGen[i];

	clock_t t;
	t = clock();

	/* std::cout << "clock start: " << t << "\n"; */

        for (int i = 0; i < PROFILING_TESTS; ++i) {
            rle(in, n, g_symbolsOut, g_countsOut);
        }

	t = clock() - t;

	std::cout << n << " BYTES compressed in " << t << " microseconds\n";
	/* std::cout << "Verifying Compression\n"; */

	/* if (verifyCompression(in, n, g_symbolsOut, g_countsOut, 10)) */
	/* 	std::cout << "Original string and Compressed String match\n"; */
	/* else */
	/* 	std::cout << "Original string and Compressed String do no match\n"; */

        /* printf("For n = %d, in time %.5f microseconds\n", n, (t || 5.0f)); */

        // also unit test, to make sure that the compression is valid.
        unitTest(in, n, rle, true);
	/* if ( n == 100 ){ */
	/* 	std::cout << sizeof(in) << "\t" << sizeof(g_symbolsOut) << "\t" << sizeof(g_countsOut) << "\n"; */
	/* 	/1* std::cout << "Input String:\n"; *1/ */
	/* 	/1* PrintArray(g_countsOut , ); *1/ */
	/* } */
    }
}

// profile some RLE implementation on the GPU.
template<typename F, typename G>
void profileGpu(F rle, G dataGen) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < NUM_TESTS; ++i) {

        int n = Tests[i];
     //   int* in = dataGen(n);
        int* in = dataGen[i];

        // transer input data to device.
        CUDA_CHECK(hipMemcpy(d_in, in, n*sizeof(int), hipMemcpyHostToDevice));

        // record.
        hipEventRecord(start);

        for (int i = 0; i < PROFILING_TESTS; ++i) {
            parleDevice(d_in, n, d_symbolsOut, d_countsOut, d_totalRuns);
        }
        hipEventRecord(stop);
        hipDeviceSynchronize();

        // also unit test, to make sure that the compression is valid.

        float ms;
        hipEventElapsedTime(&ms, start, stop);


	std::cout << n << " BYTES compressed in " << (ms / ((int)PROFILING_TESTS)) * 1000 << " microseconds\n";
        unitTest(in, n, rle, true);
        /* printf("For n = %d, in time %.5f microseconds\n", n, (ms / ((float)PROFILING_TESTS)) *1000.0f); */
    }
}

// Run many unit tests on an implementation(f) of RLE.
template<typename F>
void runTests(int a, F f) {
    printf("START UNIT TESTS\n");

    for (int i = 4; i < a; ++i) {
        for (int k = 0; k < 30; ++k) {
            int n = 2 << i;

            if (k != 0) {
                // in first test, do with nice values for 'n'
                // on the other two tests, do with slightly randomized values.
                n = (int)(n * (0.6f + 1.3f * (rand() / (float)RAND_MAX)));
            }

            int* in = generateCompressibleRandomData(n);

            unitTest(in, n, f, true);
        }
        printf("-------------------------------\n\n");
    }
}

int main(){
    srand(1000);
    CUDA_CHECK(hipSetDevice(0));

    // allocate resources on device. These arrays are used globally thoughouts the program.
    CUDA_CHECK(hipMalloc((void**)&d_backwardMask, MAX_N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_scannedBackwardMask, MAX_N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_compactedBackwardMask, (MAX_N + 1) * sizeof(int)));

    CUDA_CHECK(hipMalloc((void**)&d_in, MAX_N* sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_countsOut, MAX_N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_symbolsOut, MAX_N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_totalRuns, sizeof(int)));

    // allocate resources on the host.
    g_in = new int[MAX_N];
    g_decompressed = new int[MAX_N];
    g_symbolsOut = new int[MAX_N];
    g_countsOut = new int[MAX_N];

    int* data[NUM_TESTS];
    for (int z = 0; z < NUM_TESTS; z++){
        data[z] = generateCompressibleRandomData(Tests[z]);
    }

    /* std::cout << "\033[1;31mProfile CPU\033[0m\n"; */
    /* profileCpu(rleCpu, data); */



    std::cout << "\033[1;31mProfile GPU\033[0m\n";
    profileGpu(parleHost, data);


    // free device arrays.
    CUDA_CHECK(hipFree(d_backwardMask));
    CUDA_CHECK(hipFree(d_scannedBackwardMask));
    CUDA_CHECK(hipFree(d_compactedBackwardMask));
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_countsOut));
    CUDA_CHECK(hipFree(d_symbolsOut));
    CUDA_CHECK(hipFree(d_totalRuns));

    CUDA_CHECK(hipDeviceReset());

    // free host memory.
    delete[] g_in;
    delete[] g_decompressed;

    delete[] g_symbolsOut;
    delete[] g_countsOut;

    return 0;
}



// implementation of RLE on CPU.
int rleCpu(int *in, int n, int* symbolsOut, int* countsOut){

    if (n == 0)
        return 0; // nothing to compress!

    int outIndex = 0;
    int symbol = in[0];
    int count = 1;

    for (int i = 1; i < n; ++i) {
        if (in[i] != symbol) {
            // run is over.
            // So output run.
            symbolsOut[outIndex] = symbol;
            countsOut[outIndex] = count;
            outIndex++;

            // and start new run:
            symbol = in[i];
            count = 1;
        }
        else {
            ++count; // run is not over yet.
        }
    }

    // output last run.
    symbolsOut[outIndex] = symbol;
    countsOut[outIndex] = count;
    outIndex++;

    return outIndex;
}

// On the CPU do preparation to run parle, launch PARLE on GPU, and then transfer the result data to the CPU.
int parleHost(int *h_in, int n,
    int* h_symbolsOut,
    int* h_countsOut){

    int h_totalRuns;

    // transer input data to device.
    CUDA_CHECK(hipMemcpy(d_in, h_in, n*sizeof(int), hipMemcpyHostToDevice));

    // RUN.
    parleDevice(d_in, n, d_symbolsOut, d_countsOut, d_totalRuns);

    // transer result data to host.
    CUDA_CHECK(hipMemcpy(h_symbolsOut, d_symbolsOut, n*sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_countsOut, d_countsOut, n*sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&h_totalRuns, d_totalRuns, sizeof(int), hipMemcpyDeviceToHost));

    return h_totalRuns;
}
void fillPrefixSum(int *arr, int n, int *prefixSum)
{
    prefixSum[0] = arr[0];

    for (int i = 1; i < n; i++)
        prefixSum[i] = prefixSum[i-1] + arr[i];
    printf("check");
}

void scan(int* d_in, int* d_out, int N) {
    int *h = (int*) malloc (sizeof(int) * N);
    int *g = (int*) malloc (sizeof(int) * N);
    CUDA_CHECK(hipMemcpy(h, d_in, sizeof(int)*N, hipMemcpyDeviceToHost));
    fillPrefixSum(h, N, g);
    CUDA_CHECK(hipMemcpy(d_out, g, sizeof(int)*N, hipMemcpyHostToDevice));
}

// run parle on the GPU
void parleDevice(int *d_in, int n,
    int* d_symbolsOut,
    int* d_countsOut,
    int* d_totalRuns
    ){
    hemi::hipLaunchByPtr(maskKernel, d_in, d_backwardMask, n);
    scan(d_backwardMask, d_scannedBackwardMask, n);
    hemi::hipLaunchByPtr(compactKernel, d_in, d_scannedBackwardMask, d_compactedBackwardMask, d_totalRuns, n);
    hemi::hipLaunchByPtr(scatterKernel, d_compactedBackwardMask, d_totalRuns, d_in, d_symbolsOut, d_countsOut);
}
